#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec3 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB;
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}


// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ normal_opacity,
	const float* __restrict__ transMats,
	const float* __restrict__ colors,
	const float* __restrict__ depths,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	const float* __restrict__ dL_depths,
	float * __restrict__ dL_dtransMat,
	float3* __restrict__ dL_dmean2D,
	float* __restrict__ dL_dnormal3D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = {(float)pix.x, (float)pix.y};

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_normal_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];
	// __shared__ float collected_depths[BLOCK_SIZE];

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];

#if RENDER_AXUTILITY
	float dL_dreg;
	float dL_ddepth;
	float dL_daccum;
	float dL_dnormal2D[3];
	const int median_contributor = inside ? n_contrib[pix_id + H * W] : 0;
	float dL_dmedian_depth;
	float dL_dmax_dweight;

	if (inside) {
		dL_ddepth = dL_depths[DEPTH_OFFSET * H * W + pix_id];
		dL_daccum = dL_depths[ALPHA_OFFSET * H * W + pix_id];
		dL_dreg = dL_depths[DISTORTION_OFFSET * H * W + pix_id];
		for (int i = 0; i < 3; i++) 
			dL_dnormal2D[i] = dL_depths[(NORMAL_OFFSET + i) * H * W + pix_id];

		dL_dmedian_depth = dL_depths[MIDDEPTH_OFFSET * H * W + pix_id];
		// dL_dmax_dweight = dL_depths[MEDIAN_WEIGHT_OFFSET * H * W + pix_id];
	}

	// for compute gradient with respect to depth and normal
	float last_depth = 0;
	float last_normal[3] = { 0 };
	float accum_depth_rec = 0;
	float accum_alpha_rec = 0;
	float accum_normal_rec[3] = {0};
	// for compute gradient with respect to the distortion map
	const float final_D = inside ? final_Ts[pix_id + H * W] : 0;
	const float final_D2 = inside ? final_Ts[pix_id + 2 * H * W] : 0;
	const float final_A = 1 - T_final;
	float last_dL_dT = 0;
#endif

	if (inside){
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
	}

	float last_alpha = 0;
	float last_color[C] = { 0 };

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_normal_opacity[block.thread_rank()] = normal_opacity[coll_id];
			collected_Tu[block.thread_rank()] = {transMats[9 * coll_id+0], transMats[9 * coll_id+1], transMats[9 * coll_id+2]};
			collected_Tv[block.thread_rank()] = {transMats[9 * coll_id+3], transMats[9 * coll_id+4], transMats[9 * coll_id+5]};
			collected_Tw[block.thread_rank()] = {transMats[9 * coll_id+6], transMats[9 * coll_id+7], transMats[9 * coll_id+8]};
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];
				// collected_depths[block.thread_rank()] = depths[coll_id];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// compute ray-splat intersection as before
			// Fisrt compute two homogeneous planes, See Eq. (8)
			const float2 xy = collected_xy[j];
			const float3 Tu = collected_Tu[j];
			const float3 Tv = collected_Tv[j];
			const float3 Tw = collected_Tw[j];
			float3 k = pix.x * Tw - Tu;
			float3 l = pix.y * Tw - Tv;
			float3 p = cross(k, l);
			if (p.z == 0.0) continue;
			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y); 
			float2 d = {xy.x - pixf.x, xy.y - pixf.y};
			float rho2d = FilterInvSquare * (d.x * d.x + d.y * d.y); 

			// compute intersection and depth
			float rho = min(rho3d, rho2d);
			float c_d = (rho3d <= rho2d) ? (s.x * Tw.x + s.y * Tw.y) + Tw.z : Tw.z; 
			if (c_d < near_n) continue;
			float4 nor_o = collected_normal_opacity[j];
			float normal[3] = {nor_o.x, nor_o.y, nor_o.z};
			float opa = nor_o.w;

			// accumulations

			float power = -0.5f * rho;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			const float alpha = min(0.99f, opa * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;
			const float w = alpha * T;
			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}

			float dL_dz = 0.0f;
			float dL_dweight = 0;
#if RENDER_AXUTILITY
			const float m_d = far_n / (far_n - near_n) * (1 - near_n / c_d);
			const float dmd_dd = (far_n * near_n) / ((far_n - near_n) * c_d * c_d);
			if (contributor == median_contributor-1) {
				dL_dz += dL_dmedian_depth;
				// dL_dweight += dL_dmax_dweight;
			}
#if DETACH_WEIGHT 
			// if not detached weight, sometimes 
			// it will bia toward creating extragated 2D Gaussians near front
			dL_dweight += 0;
#else
			dL_dweight += (final_D2 + m_d * m_d * final_A - 2 * m_d * final_D) * dL_dreg;
#endif
			dL_dalpha += dL_dweight - last_dL_dT;
			// propagate the current weight W_{i} to next weight W_{i-1}
			last_dL_dT = dL_dweight * alpha + (1 - alpha) * last_dL_dT;
			const float dL_dmd = 2.0f * (T * alpha) * (m_d * final_A - final_D) * dL_dreg;
			dL_dz += dL_dmd * dmd_dd;

			// Propagate gradients w.r.t ray-splat depths
			accum_depth_rec = last_alpha * last_depth + (1.f - last_alpha) * accum_depth_rec;
			last_depth = c_d;
			dL_dalpha += (c_d - accum_depth_rec) * dL_ddepth;
			// Propagate gradients w.r.t. color ray-splat alphas
			accum_alpha_rec = last_alpha * 1.0 + (1.f - last_alpha) * accum_alpha_rec;
			dL_dalpha += (1 - accum_alpha_rec) * dL_daccum;

			// Propagate gradients to per-Gaussian normals
			for (int ch = 0; ch < 3; ch++) {
				accum_normal_rec[ch] = last_alpha * last_normal[ch] + (1.f - last_alpha) * accum_normal_rec[ch];
				last_normal[ch] = normal[ch];
				dL_dalpha += (normal[ch] - accum_normal_rec[ch]) * dL_dnormal2D[ch];
				atomicAdd((&dL_dnormal3D[global_id * 3 + ch]), alpha * T * dL_dnormal2D[ch]);
			}
#endif

			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;


			// Helpful reusable temporary variables
			const float dL_dG = nor_o.w * dL_dalpha;
#if RENDER_AXUTILITY
			dL_dz += alpha * T * dL_ddepth; 
#endif

			if (rho3d <= rho2d) {
				// Update gradients w.r.t. covariance of Gaussian 3x3 (T)
				const float2 dL_ds = {
					dL_dG * -G * s.x + dL_dz * Tw.x,
					dL_dG * -G * s.y + dL_dz * Tw.y
				};
				const float3 dz_dTw = {s.x, s.y, 1.0};
				const float dsx_pz = dL_ds.x / p.z;
				const float dsy_pz = dL_ds.y / p.z;
				const float3 dL_dp = {dsx_pz, dsy_pz, -(dsx_pz * s.x + dsy_pz * s.y)};
				const float3 dL_dk = cross(l, dL_dp);
				const float3 dL_dl = cross(dL_dp, k);

				const float3 dL_dTu = {-dL_dk.x, -dL_dk.y, -dL_dk.z};
				const float3 dL_dTv = {-dL_dl.x, -dL_dl.y, -dL_dl.z};
				const float3 dL_dTw = {
					pixf.x * dL_dk.x + pixf.y * dL_dl.x + dL_dz * dz_dTw.x, 
					pixf.x * dL_dk.y + pixf.y * dL_dl.y + dL_dz * dz_dTw.y, 
					pixf.x * dL_dk.z + pixf.y * dL_dl.z + dL_dz * dz_dTw.z};


				// Update gradients w.r.t. 3D covariance (3x3 matrix)
				atomicAdd(&dL_dtransMat[global_id * 9 + 0],  dL_dTu.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 1],  dL_dTu.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 2],  dL_dTu.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 3],  dL_dTv.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 4],  dL_dTv.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 5],  dL_dTv.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 6],  dL_dTw.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 7],  dL_dTw.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 8],  dL_dTw.z);
			} else {
				// // Update gradients w.r.t. center of Gaussian 2D mean position
				const float dG_ddelx = -G * FilterInvSquare * d.x;
				const float dG_ddely = -G * FilterInvSquare * d.y;
				atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx); // not scaled
				atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely); // not scaled
				atomicAdd(&dL_dtransMat[global_id * 9 + 8],  dL_dz); // propagate depth loss
			}

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dalpha);
		}
	}
}


__device__ void compute_transmat_aabb(
	int idx, 
	const float* Ts_precomp,
	const float3* p_origs, 
	const glm::vec2* scales, 
	const glm::vec4* rots, 
	const float* projmatrix, 
	const float* viewmatrix, 
	const int W, const int H, 
	const float3* dL_dnormals,
	const float3* dL_dmean2Ds, 
	float* dL_dTs, 
	glm::vec3* dL_dmeans, 
	glm::vec2* dL_dscales,
	 glm::vec4* dL_drots)
{
	glm::mat3 T;
	float3 normal;
	glm::mat3x4 P;
	glm::mat3 R;
	glm::mat3 S;
	float3 p_orig;
	glm::vec4 rot;
	glm::vec2 scale;
	
	// Get transformation matrix of the Gaussian
	if (Ts_precomp != nullptr) {
		T = glm::mat3(
			Ts_precomp[idx * 9 + 0], Ts_precomp[idx * 9 + 1], Ts_precomp[idx * 9 + 2],
			Ts_precomp[idx * 9 + 3], Ts_precomp[idx * 9 + 4], Ts_precomp[idx * 9 + 5],
			Ts_precomp[idx * 9 + 6], Ts_precomp[idx * 9 + 7], Ts_precomp[idx * 9 + 8]
		);
		normal = {0.0, 0.0, 0.0};
	} else {
		p_orig = p_origs[idx];
		rot = rots[idx];
		scale = scales[idx];
		R = quat_to_rotmat(rot);
		S = scale_to_mat(scale, 1.0f);
		
		glm::mat3 L = R * S;
		glm::mat3x4 M = glm::mat3x4(
			glm::vec4(L[0], 0.0),
			glm::vec4(L[1], 0.0),
			glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1)
		);

		glm::mat4 world2ndc = glm::mat4(
			projmatrix[0], projmatrix[4], projmatrix[8], projmatrix[12],
			projmatrix[1], projmatrix[5], projmatrix[9], projmatrix[13],
			projmatrix[2], projmatrix[6], projmatrix[10], projmatrix[14],
			projmatrix[3], projmatrix[7], projmatrix[11], projmatrix[15]
		);

		glm::mat3x4 ndc2pix = glm::mat3x4(
			glm::vec4(float(W) / 2.0, 0.0, 0.0, float(W-1) / 2.0),
			glm::vec4(0.0, float(H) / 2.0, 0.0, float(H-1) / 2.0),
			glm::vec4(0.0, 0.0, 0.0, 1.0)
		);

		P = world2ndc * ndc2pix;
		T = glm::transpose(M) * P;
		normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);
	}

	// Update gradients w.r.t. transformation matrix of the Gaussian
	glm::mat3 dL_dT = glm::mat3(
		dL_dTs[idx*9+0], dL_dTs[idx*9+1], dL_dTs[idx*9+2],
		dL_dTs[idx*9+3], dL_dTs[idx*9+4], dL_dTs[idx*9+5],
		dL_dTs[idx*9+6], dL_dTs[idx*9+7], dL_dTs[idx*9+8]
	);
	float3 dL_dmean2D = dL_dmean2Ds[idx];
	if(dL_dmean2D.x != 0 || dL_dmean2D.y != 0)
	{
		const float distance = T[2].x * T[2].x + T[2].y * T[2].y - T[2].z * T[2].z;
		const float f = 1 / (distance);
		const float dpx_dT00 =  f * T[2].x;
		const float dpx_dT01 =  f * T[2].y;
		const float dpx_dT02 = -f * T[2].z;
		const float dpy_dT10 =  f * T[2].x;
		const float dpy_dT11 =  f * T[2].y;
		const float dpy_dT12 = -f * T[2].z;
		const float dpx_dT30 =  T[0].x * (f - 2 * f * f * T[2].x * T[2].x);
		const float dpx_dT31 =  T[0].y * (f - 2 * f * f * T[2].y * T[2].y);
		const float dpx_dT32 = -T[0].z * (f + 2 * f * f * T[2].z * T[2].z);
		const float dpy_dT30 =  T[1].x * (f - 2 * f * f * T[2].x * T[2].x);
		const float dpy_dT31 =  T[1].y * (f - 2 * f * f * T[2].y * T[2].y);
		const float dpy_dT32 = -T[1].z * (f + 2 * f * f * T[2].z * T[2].z);

		dL_dT[0].x += dL_dmean2D.x * dpx_dT00;
		dL_dT[0].y += dL_dmean2D.x * dpx_dT01;
		dL_dT[0].z += dL_dmean2D.x * dpx_dT02;
		dL_dT[1].x += dL_dmean2D.y * dpy_dT10;
		dL_dT[1].y += dL_dmean2D.y * dpy_dT11;
		dL_dT[1].z += dL_dmean2D.y * dpy_dT12;
		dL_dT[2].x += dL_dmean2D.x * dpx_dT30 + dL_dmean2D.y * dpy_dT30;
		dL_dT[2].y += dL_dmean2D.x * dpx_dT31 + dL_dmean2D.y * dpy_dT31;
		dL_dT[2].z += dL_dmean2D.x * dpx_dT32 + dL_dmean2D.y * dpy_dT32;

		if (Ts_precomp != nullptr) {
			dL_dTs[idx * 9 + 0] = dL_dT[0].x;
			dL_dTs[idx * 9 + 1] = dL_dT[0].y;
			dL_dTs[idx * 9 + 2] = dL_dT[0].z;
			dL_dTs[idx * 9 + 3] = dL_dT[1].x;
			dL_dTs[idx * 9 + 4] = dL_dT[1].y;
			dL_dTs[idx * 9 + 5] = dL_dT[1].z;
			dL_dTs[idx * 9 + 6] = dL_dT[2].x;
			dL_dTs[idx * 9 + 7] = dL_dT[2].y;
			dL_dTs[idx * 9 + 8] = dL_dT[2].z;
			return;
		}
	}
	
	if (Ts_precomp != nullptr) return;

	// Update gradients w.r.t. scaling, rotation, position of the Gaussian
	glm::mat3x4 dL_dM = P * glm::transpose(dL_dT);
	float3 dL_dtn = transformVec4x3Transpose(dL_dnormals[idx], viewmatrix);
#if DUAL_VISIABLE
	float multiplier = normal.z < 0 ? 1: -1;
	dL_dtn = multiplier * dL_dtn;
#endif
	glm::mat3 dL_dRS = glm::mat3(
		glm::vec3(dL_dM[0]),
		glm::vec3(dL_dM[1]),
		glm::vec3(dL_dtn.x, dL_dtn.y, dL_dtn.z)
	);

	glm::mat3 dL_dR = glm::mat3(
		dL_dRS[0] * glm::vec3(scale.x),
		dL_dRS[1] * glm::vec3(scale.y),
		dL_dRS[2]);
	
	dL_drots[idx] = quat_to_rotmat_vjp(rot, dL_dR);
	dL_dscales[idx] = glm::vec2(
		(float)glm::dot(dL_dRS[0], R[0]),
		(float)glm::dot(dL_dRS[1], R[1])
	);
	dL_dmeans[idx] = glm::vec3(dL_dM[2]);
}

template<int C>
__global__ void preprocessCUDA(
	int P, int D, int M,
	const float3* means3D,
	const float* transMats,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, 
	const float focal_y,
	const float tan_fovx,
	const float tan_fovy,
	const glm::vec3* campos, 
	// grad input
	float* dL_dtransMats,
	const float* dL_dnormal3Ds,
	float* dL_dcolors,
	float* dL_dshs,
	float3* dL_dmean2Ds,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	const int W = int(focal_x * tan_fovx * 2);
	const int H = int(focal_y * tan_fovy * 2);
	const float * Ts_precomp = (scales) ? nullptr : transMats;
	compute_transmat_aabb(
		idx, 
		Ts_precomp,
		means3D, scales, rotations, 
		projmatrix, viewmatrix, W, H, 
		(float3*)dL_dnormal3Ds, 
		dL_dmean2Ds,
		(dL_dtransMats), 
		dL_dmean3Ds, 
		dL_dscales, 
		dL_drots
	);

	if (shs)
		computeColorFromSH(idx, D, M, (glm::vec3*)means3D, *campos, shs, clamped, (glm::vec3*)dL_dcolors, (glm::vec3*)dL_dmean3Ds, (glm::vec3*)dL_dshs);
	
	// hack the gradient here for densitification
	float depth = transMats[idx * 9 + 8];
	dL_dmean2Ds[idx].x = dL_dtransMats[idx * 9 + 2] * depth * 0.5 * float(W); // to ndc 
	dL_dmean2Ds[idx].y = dL_dtransMats[idx * 9 + 5] * depth * 0.5 * float(H); // to ndc
}


void BACKWARD::preprocess(
	int P, int D, int M,
	const float3* means3D,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* transMats,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, const float focal_y,
	const float tan_fovx, const float tan_fovy,
	const glm::vec3* campos, 
	float3* dL_dmean2Ds,
	const float* dL_dnormal3Ds,
	float* dL_dtransMats,
	float* dL_dcolors,
	float* dL_dshs,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots)
{	
	preprocessCUDA<NUM_CHANNELS><< <(P + 255) / 256, 256 >> > (
		P, D, M,
		(float3*)means3D,
		transMats,
		radii,
		shs,
		clamped,
		(glm::vec2*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		viewmatrix,
		projmatrix,
		focal_x, 
		focal_y,
		tan_fovx,
		tan_fovy,
		campos,	
		dL_dtransMats,
		dL_dnormal3Ds,
		dL_dcolors,
		dL_dshs,
		dL_dmean2Ds,
		dL_dmean3Ds,
		dL_dscales,
		dL_drots
	);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* bg_color,
	const float2* means2D,
	const float4* normal_opacity,
	const float* colors,
	const float* transMats,
	const float* depths,
	const float* final_Ts,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	const float* dL_depths,
	float * dL_dtransMat,
	float3* dL_dmean2D,
	float* dL_dnormal3D,
	float* dL_dopacity,
	float* dL_dcolors)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		focal_x, focal_y,
		bg_color,
		means2D,
		normal_opacity,
		transMats,
		colors,
		depths,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_depths,
		dL_dtransMat,
		dL_dmean2D,
		dL_dnormal3D,
		dL_dopacity,
		dL_dcolors
		);
}
